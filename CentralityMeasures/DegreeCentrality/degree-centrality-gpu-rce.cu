
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdlib.h>

__global__ void dc_kernel(int n, int *R, int *C, double *dc);

int main(int argc, char const *argv[]) {
    
    /* File che rappresentano il grafo in formato RCE */
    FILE *R = fopen("data/row_offsets.dat", "r");
    FILE *C = fopen("data/column_indices.dat", "r");

    int n, r_size, c_size;
    int *h_r, *h_c;
    int *d_r, *d_c;
    double *h_dc, *d_dc;

    /* Input: numero di nodi e archi del grafo */
    printf("Inserire numero di nodi: ");
    scanf("%d", &n);
    r_size = n + 1;

    printf("Inserire numero di archi: ");
    scanf("%d", &c_size);

    /* Allocazione strutture dati host */
    h_r = (int*)malloc(r_size * sizeof(int));
    h_c = (int*)malloc(c_size * sizeof(int));
    h_dc = (double*)malloc(n * sizeof(int));

    /* Allocazione strutture dati device */
    hipMalloc((void **) &d_r, r_size * sizeof(int));
    hipMalloc((void **) &d_c, c_size * sizeof(int));
    hipMalloc((void **) &d_dc, n * sizeof(double));

    /* Leggo da file il columns indices ed il row offsets array */
    for (int i = 0; i < r_size; i++) {
        fscanf(R, "%d\n", &h_r[i]);
    }
    for (int i = 0; i < c_size; i++) {
        fscanf(C, "%d\n", &h_c[i]);
    }

    /* Copia da host a device */
    hipMemcpy(d_r, h_r, r_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, c_size * sizeof(int), hipMemcpyHostToDevice);

    /* Configurazione del Kernel */
    dim3 blockDim(64);
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x);

    /* Invocazione del kernel */
    dc_kernel<<<gridDim, blockDim>>>(n, d_r, d_c, d_dc);

    /* Copia dei risultati da device a host */
    hipMemcpy(h_dc, d_dc, n * sizeof(double), hipMemcpyDeviceToHost);

    /* Stampa dei risultati */
    printf("Degree Centrality:\n");
    for (int i = 0; i < n; i++) {
        printf("Score %d: %f\n", i+1, h_dc[i]);
    }


    return 0;
}

__global__ void dc_kernel(int n, int *R, int *C, double *dc) {
    int idx = threadIdx.x;

    if (idx < n) {
        dc[idx] = (double) (R[idx+1] - R[idx]) / (n - 1);
    }
}